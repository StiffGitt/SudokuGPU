#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <iomanip>

using namespace std::chrono;

#define BOARD_FILE "board.txt"
#define N 9
#define R 81
#define C 90
#define S 99
#define BSIZE 108
#define ITERATIONS 18
#define BLOCKS 256
#define THREADS 512
#define MEMSIZE 28
typedef unsigned int Uint;
#define SET_BIT(a,b) a |= ((Uint)1 << b);
#define CLEAR_BIT(a, b) a &= ~((Uint)1 << b);
#define CHECK_BIT(a, b) (bool)((a >> b) & (Uint)1)
#define GET_ROW(i) (R + i / N);
#define GET_COLUMN(i) (int)(C + i % N);
#define GET_SUBBOARD(i) (int)(S + ((i / N) / 3) * 3 + ((i % N) / 3));

Uint* solveSudoku(Uint* board);
void printBoard(Uint* board, bool printAdditional);
Uint* initializeBoard(Uint* board);
void initializeRows(Uint* board);
void initializeColumns(Uint* board);
void initializeSubboards(Uint* board);
Uint* loadBoard();
bool checkSolution(Uint* solution, Uint* board);
__global__ void cudaDoBFS(Uint* oldBoard, Uint* newBoard, int boardsCount, int* lastBoard, int* empties, int isFinal);
__global__ void cudaDoDFS(Uint* board, int boardsCount, int* empties, int* outStatus, Uint* solution);

int main()
{
    Uint* board = loadBoard();

    std::cout << "Input:";

    printBoard(board, false);

    auto clockStart = high_resolution_clock::now();

    Uint* solution = solveSudoku(board);

    auto clockStop = high_resolution_clock::now();
    std::cout << "\nFull Time:    " << std::setw(7) << 0.001 * duration_cast<microseconds>(clockStop - clockStart).count() << " milisec\n";

    std::cout << "\nSolution: ";

    printBoard(solution, false);

    if (checkSolution(solution, board))
        std::cout << "\nCORRECT SOLUTION!!";
    else
        std::cout << "\nINCORRECT SOLUTION!!";

    return 0;
}

Uint* solveSudoku(Uint *inBoard)
{
    std::chrono::steady_clock::time_point clockStart, clockStop;

    clockStart = high_resolution_clock::now();
    Uint* board = initializeBoard(inBoard);

    const int boardMem = pow(2, MEMSIZE);
    Uint* board1, *board2, *solution;
    int* lastBoard, *empties, *emptiesCount, *outStatus;

    hipMalloc(&board1, boardMem * sizeof(Uint));
    hipMalloc(&board2, boardMem * sizeof(Uint));
    hipMalloc(&empties, boardMem * sizeof(Uint));
    hipMalloc(&lastBoard, sizeof(int));
    hipMalloc(&outStatus, sizeof(int));
    hipMalloc(&solution, N * N * sizeof(Uint));

    hipMemset(board1, 0, boardMem * sizeof(Uint));
    hipMemset(board2, 0, boardMem * sizeof(Uint));
    hipMemset(empties, -1, boardMem * sizeof(Uint));
    hipMemset(outStatus, 0, sizeof(int));
    hipMemset(solution, 0, N * N * sizeof(Uint));

    hipMemcpy(board1, board, BSIZE * sizeof(Uint), hipMemcpyHostToDevice);
    int boardsCount = 1;
    int isFinal = 0;
    int itDone = 0;

    clockStop = high_resolution_clock::now();
    std::cout << "\nMemory allocation:    " << std::setw(7) << 0.001 * duration_cast<microseconds>(clockStop - clockStart).count() << " milisec\n\n";
    clockStart = high_resolution_clock::now();
    for (int it = 0; it < ITERATIONS; it++)
    {
        hipMemset(lastBoard, 0, sizeof(int));
        
        if (it == ITERATIONS - 1)
            isFinal = 1;

        if(it % 2 == 0)
            cudaDoBFS<<< BLOCKS, THREADS >>>(board1, board2, boardsCount, lastBoard, empties, isFinal);
        else
            cudaDoBFS<<< BLOCKS, THREADS >>>(board2, board1, boardsCount, lastBoard, empties, isFinal);

        hipMemcpy(&boardsCount, lastBoard, sizeof(int), hipMemcpyDeviceToHost);
        
         printf("boards count after it %d: %d\n", it, boardsCount);

         if (boardsCount > (boardMem / BSIZE))
         {
             std::cout << "\n memory overflow, bfs needs more memory, exiting... \n";
             return inBoard;
         }
         itDone = it + 1;
    }

    Uint* bfsBoard = (itDone % 2 == 0)? board1 : board2;

    hipDeviceSynchronize();
    clockStop = high_resolution_clock::now();
    std::cout << "\nBFS:    " << std::setw(7) << 0.001 * duration_cast<microseconds>(clockStop - clockStart).count() << " milisec\n";

    /*hipMemcpy(board, bfsBoard, BSIZE * sizeof(Uint), hipMemcpyDeviceToHost);
    printBoard(board, true);*/

    /*int tab[N * N];
    hipMemcpy(tab, empties, N * N * sizeof(int), hipMemcpyDeviceToHost);
    printBoard(tab, false);*/

    clockStart = high_resolution_clock::now();

    cudaDoDFS<<< BLOCKS, THREADS >>>(bfsBoard, boardsCount, empties, outStatus, solution);

    hipDeviceSynchronize();
    clockStop = high_resolution_clock::now();
    std::cout << "\nDFS:    " << std::setw(7) << 0.001 * duration_cast<microseconds>(clockStop - clockStart).count() << " milisec\n";

    hipMemcpy(board, solution, N * N * sizeof(Uint), hipMemcpyDeviceToHost);

    hipFree(board1);
    hipFree(board2);
    hipFree(lastBoard);
    hipFree(empties);
    hipFree(outStatus);
    hipFree(solution);
    
    return board;
}

void printBoard(Uint* board, bool printAdditional)
{
    std::cout << "\n\n";
    for (int i = 0; i < N; i++)
    {
        if (i == 3 || i == 6)
            std::cout << "-------------------\n";
        for (int j = 0; j < N; j++)
        {
            if (j == 3 || j == 6)
                std::cout << "|";
            std::cout << board[i * N + j] << " ";
        }
        std::cout << "\n";
    }
    if (!printAdditional)
        return;
    std::cout << "\n" << "rows: " << "\n";
    for (int i = 0; i < N; i++)
    {
        std::cout << board[R + i] << " ";
    }
    std::cout << "\n" << "columns: " << "\n";
    for (int i = 0; i < N; i++)
    {
        std::cout << board[C + i] << " ";
    }
    std::cout << "\n" << "subboards: " << "\n";
    for (int i = 0; i < N; i++)
    {
        std::cout << board[S + i] << " ";
    }
}

Uint* initializeBoard(Uint* inBoard)
{
    Uint *board = (Uint*)malloc(BSIZE * sizeof(Uint));
    std::copy(inBoard, inBoard + N * N, board);
    initializeRows(board);
    initializeColumns(board);
    initializeSubboards(board);

    return board;
}

void initializeRows(Uint* board)
{
    for (int r = 0; r < N; r++)
    {
        board[R + r] = (Uint)pow(2, N) - 1;
        for (int i = 0; i < N; i++)
        {
            if (board[r * N + i] > 0)
                CLEAR_BIT(board[R + r], (board[r * N + i] - 1));
        }
    }
}

void initializeColumns(Uint* board)
{
    for (int c = 0; c < N; c++)
    {
        board[C + c] = (Uint)pow(2, N) - 1;
        for (int i = 0; i < N; i++)
        {
            if (board[i * N + c] > 0)
                CLEAR_BIT(board[C + c], (board[i * N + c] - 1));
        }
    }
}

void initializeSubboards(Uint* board)
{
    for (int s = 0; s < N; s++)
    {
        board[S + s] = (Uint)pow(2, N) - 1;
        int start = ((s / 3) * 3) * N + ((s % 3) * 3);
        for (int i = 0; i < N; i++)
        {
            int r = i / 3;
            int c = i % 3;
            if (board[start + r * N + c] > 0)
                CLEAR_BIT(board[S + s], (board[start + r * N + c] - 1));
        }
    }
}

Uint* loadBoard()
{
    FILE* boardFile = fopen(BOARD_FILE, "r");
    Uint* board = (Uint*)malloc(N * N * sizeof(Uint));
    char c;

    if (boardFile == NULL)
    {
        printf("boardFile error\n");
        return board;
    }

    for (int i = 0; i < N; i++) 
    {
        for (int j = 0; j < N; j++) 
        {
            if (!fscanf(boardFile, "%c\n", &c)) 
            {
                printf("boardFile error\n");
                return board;
            }

            if (c >= '1' && c <= '9') {
                board[i * N + j] = (Uint)(c - '0');
            }
            else {
                board[i * N + j] = 0;
            }
        }
    }
    return board;
}

void clearChecked(bool* checked)
{
    for (int i = 0; i < N; i++)
    {
        checked[i] = false;
    }
}

bool checkSolution(Uint* solution, Uint* board)
{
    bool isValid = true;
    for (int i = 0; i < N * N; i++)
    {
        if (board[i] > 0 && solution[i] != board[i])
            isValid = false;
        if (solution[i] == 0)
            return false;
    }
    bool checked[N];
    for (int r = 0; r < N; r++)
    {
        clearChecked(checked);
        for (int i = 0; i < N; i++)
        {
            if (checked[solution[r * N + i] - 1] == true)
                isValid = false;
            checked[solution[r * N + i] - 1] = true;
        }
    }
    for (int c = 0; c < N; c++)
    {
        clearChecked(checked);
        for (int i = 0; i < N; i++)
        {
            if (checked[solution[i * N + c] - 1] == true)
                isValid = false;
            checked[solution[i * N + c] - 1] = true;
        }
    }

    for (int s = 0; s < N; s++)
    {
        clearChecked(checked);
        int start = ((s / 3) * 3) * N + ((s % 3) * 3);
        for (int i = 0; i < N; i++)
        {
            int r = i / 3;
            int c = i % 3;
            if (checked[solution[start + r * N + c] - 1] == true)
                isValid = false;
            checked[solution[start + r * N + c] - 1] = true;
        }
    }

    return isValid;
}

__global__ void cudaDoBFS(Uint* oldBoard, Uint* newBoard, int boardsCount, int* lastBoard, int* empties, int isFinal)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < boardsCount)
    {
        int boardBegin = idx * BSIZE;
        int i = boardBegin;
        while (i < boardBegin + N * N)
        {
            if (oldBoard[i] == 0)
                break;
            i++;
        }
        if (i == boardBegin + N * N)
            return;

        int r = boardBegin + GET_ROW((i - boardBegin));
        int c = boardBegin + GET_COLUMN((i - boardBegin));
        int s = boardBegin + GET_SUBBOARD((i - boardBegin));
        for (int number = 1; number <= N; number++)
        {
            if (CHECK_BIT(oldBoard[r], (number - 1)) && CHECK_BIT(oldBoard[c], (number - 1)) && CHECK_BIT(oldBoard[s], (number - 1)))
            {
                int curBoard = atomicAdd(lastBoard, 1);
                int newBegin = curBoard * BSIZE;
                int emptiesIdx = curBoard * N * N;
                for (int j = 0; j < BSIZE; j++)
                {
                    newBoard[newBegin + j] = oldBoard[boardBegin + j];
                    if (isFinal == 1 && j < N * N && oldBoard[boardBegin + j] == 0 && boardBegin + j != i)
                    {
                        empties[emptiesIdx] = j;
                        emptiesIdx++;
                    }
                }
                newBoard[newBegin + i - boardBegin] = number;
                CLEAR_BIT(newBoard[newBegin + r - boardBegin], (number - 1));
                CLEAR_BIT(newBoard[newBegin + c - boardBegin], (number - 1));
                CLEAR_BIT(newBoard[newBegin + s - boardBegin], (number - 1));
            }
        }
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void cudaDoDFS(Uint* board, int boardsCount, int* empties, int* outStatus, Uint* solution)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while ((*outStatus) == 0 && idx < boardsCount)
    {
        int boardBegin = idx * BSIZE;
        int emptiesBegin = idx * N * N;
        int emptiesIdx = emptiesBegin;

        while (empties[emptiesIdx] >= 0)
        {
            int boardIdx = boardBegin + empties[emptiesIdx];
            int r = boardBegin + GET_ROW((empties[emptiesIdx]));
            int c = boardBegin + GET_COLUMN((empties[emptiesIdx]));
            int s = boardBegin + GET_SUBBOARD((empties[emptiesIdx]));

            board[boardIdx]++;
            Uint number = board[boardIdx];
            if (CHECK_BIT(board[r], (number - 1)) && CHECK_BIT(board[c], (number - 1)) && CHECK_BIT(board[s], (number - 1)))
            {
                CLEAR_BIT(board[r], (number - 1));
                CLEAR_BIT(board[c], (number - 1));
                CLEAR_BIT(board[s], (number - 1));
                emptiesIdx++;
            }
            else
            {
                if (board[boardIdx] >= 9)
                {
                    board[boardIdx] = 0;
                    emptiesIdx--;
                    if (emptiesIdx < emptiesBegin)
                        break;
                    boardIdx = boardBegin + empties[emptiesIdx];
                    r = boardBegin + GET_ROW((empties[emptiesIdx]));
                    c = boardBegin + GET_COLUMN((empties[emptiesIdx]));
                    s = boardBegin + GET_SUBBOARD((empties[emptiesIdx]));
                    number = board[boardIdx];

                    SET_BIT(board[r], (number - 1));
                    SET_BIT(board[c], (number - 1));
                    SET_BIT(board[s], (number - 1));

                    if (r == 81)
                    {
                        board[boardBegin + BSIZE - 1] = emptiesIdx;
                    }
                }
            }
        }

        if (empties[emptiesIdx] < 0 && emptiesIdx > emptiesBegin)
        {
            *outStatus = 1;
            for (int i = 0; i < N * N; i++)
            {
                solution[i] = board[boardBegin + i];
            }
        }

        idx += blockDim.x * gridDim.x;
    }
}

//void printBoard(int* board, bool printAdditional)
//{
//    std::cout << "\n\n";
//    for (int i = 0; i < N; i++)
//    {
//        if (i == 3 || i == 6)
//            std::cout << "-------------------\n";
//        for (int j = 0; j < N; j++)
//        {
//            if (j == 3 || j == 6)
//                std::cout << "|";
//            std::cout << board[i * N + j] << " ";
//        }
//        std::cout << "\n";
//    }
//    if (!printAdditional)
//        return;
//    std::cout << "\n" << "rows: " << "\n";
//    for (int i = 0; i < N; i++)
//    {
//        std::cout << board[R + i] << " ";
//    }
//    std::cout << "\n" << "columns: " << "\n";
//    for (int i = 0; i < N; i++)
//    {
//        std::cout << board[C + i] << " ";
//    }
//    std::cout << "\n" << "subboards: " << "\n";
//    for (int i = 0; i < N; i++)
//    {
//        std::cout << board[S + i] << " ";
//    }
//}

﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

#define BOARD_FILE "board.txt"
#define N 9
#define R 81
#define C 90
#define S 99
#define BSIZE 108
#define ITERATIONS 20
#define BLOCKS 256
#define THREADS 512
typedef unsigned int Uint;
#define CLEAR_BIT(a, b) a &= ~((Uint)1 << b);
#define CHECK_BIT(a, b) (bool)((a >> b) & (Uint)1)
#define GET_ROW(i) (R + i / N);
#define GET_COLUMN(i) (int)(C + i % N);
#define GET_SUBBOARD(i) (int)(S + ((i / N) / 3) * 3 + ((i % N) / 3));

Uint* solveSudoku(Uint* board);
void printBoard(Uint* board);
Uint* initializeBoard(Uint* board);
void initializeRows(Uint* board);
void initializeColumns(Uint* board);
void initializeSubboards(Uint* board);
Uint* loadBoard();

__global__ void cudaBFS(Uint* oldBoard, Uint* newBoard, int boardsCount, int *lastBoard)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while(idx < boardsCount)
    {
        int boardBegin = idx * BSIZE;
        int i = boardBegin; 
        while(i < boardBegin + N * N)
        {
            if(oldBoard[i] == 0)
                break;
            i++;
        }
        if (i == boardBegin + N * N)
            return;

        int r = boardBegin + GET_ROW((i - boardBegin));
        int c = boardBegin + GET_COLUMN((i - boardBegin));
        int s = boardBegin + GET_SUBBOARD((i - boardBegin));
        for (int number = 1; number <= N; number++)
        {
            if (CHECK_BIT(oldBoard[r], (number - 1)) && CHECK_BIT(oldBoard[c], (number - 1)) && CHECK_BIT(oldBoard[s], (number - 1)))
            {
                int curBoard = atomicAdd(lastBoard, 1);
                int newBegin = curBoard * BSIZE;
                for (int j = 0; j < BSIZE; j++)
                {
                    newBoard[newBegin + j] = oldBoard[boardBegin + j];
                }
                newBoard[newBegin + i - boardBegin] = number;
                CLEAR_BIT(newBoard[newBegin + r - boardBegin], (number - 1));
                CLEAR_BIT(newBoard[newBegin + c - boardBegin], (number - 1));
                CLEAR_BIT(newBoard[newBegin + s - boardBegin], (number - 1));
            }
        }
        idx += blockDim.x * gridDim.x;
    }
}

int main()
{
    /*Uint board[]{
        0, 4, 0, 0, 0, 0, 1, 7, 9,
        0, 0, 2, 0, 0, 8, 0, 5, 4,
        0, 0, 6, 0, 0, 5, 0, 0, 8,
        0, 8, 0, 0, 7, 0, 9, 1, 0,
        0, 5, 0, 0, 9, 0, 0, 3, 0,
        0, 1, 9, 0, 6, 0, 0, 4, 0,
        3, 0, 0, 4, 0, 0, 7, 0, 0,
        5, 7, 0, 1, 0, 0, 2, 0, 0,
        9, 2, 8, 0, 0, 0, 0, 6, 0 };*/
    Uint* board = loadBoard();
    solveSudoku(board);
    return 0;
}

Uint* solveSudoku(Uint *inBoard)
{
    Uint* board = initializeBoard(inBoard);
    printBoard(board);

    const int boardMem = pow(2, 26);
    Uint* board1, *board2;
    int* lastBoard;

    hipMalloc(&board1, boardMem * sizeof(Uint));
    hipMalloc(&board2, boardMem * sizeof(Uint));
    hipMalloc(&lastBoard, sizeof(int));

    hipMemset(board1, 0, boardMem * sizeof(Uint));
    hipMemset(board2, 0, boardMem * sizeof(Uint));

    hipMemcpy(board1, board, BSIZE * sizeof(Uint), hipMemcpyHostToDevice);
    int boardsCount = 1;
    
    for (int it = 0; it < ITERATIONS; it++)
    {
        hipMemset(lastBoard, 0, sizeof(int));
        //sie pierdoli
        if(it % 2 == 0)
            cudaBFS<<< BLOCKS, THREADS >>>(board1, board2, boardsCount, lastBoard);
        else
            cudaBFS<<< BLOCKS, THREADS >>> (board2, board1, boardsCount, lastBoard);

        hipMemcpy(&boardsCount, lastBoard, sizeof(int), hipMemcpyDeviceToHost);
        
         printf("total boards after an iteration %d: %d\n", it, boardsCount);
    }

    Uint* bfsBoard = (ITERATIONS % 2 == 0)? board1 : board2;
    hipMemcpy(board, bfsBoard, BSIZE * sizeof(Uint), hipMemcpyDeviceToHost);
    printBoard(board);

    hipFree(board1);
    hipFree(board2);
    hipFree(lastBoard);
    return board;
}

void printBoard(Uint* board)
{
    std::cout << "\n\n";
    for (int i = 0; i < N; i++)
    {
        if (i == 3 || i == 6)
            std::cout << "-------------------\n";
        for (int j = 0; j < N; j++)
        {
            if (j == 3 || j == 6)
                std::cout << "|";
            std::cout << board[i * N + j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n" << "rows: " << "\n";
    for (int i = 0; i < N; i++)
    {
        std::cout << board[R + i] << " ";
    }
    std::cout << "\n" << "columns: " << "\n";
    for (int i = 0; i < N; i++)
    {
        std::cout << board[C + i] << " ";
    }
    std::cout << "\n" << "subboards: " << "\n";
    for (int i = 0; i < N; i++)
    {
        std::cout << board[S + i] << " ";
    }
}

Uint* initializeBoard(Uint* inBoard)
{
    Uint *board = (Uint*)malloc(BSIZE * sizeof(Uint));
    std::copy(inBoard, inBoard + N * N, board);
    initializeRows(board);
    initializeColumns(board);
    initializeSubboards(board);

    return board;
}
void initializeRows(Uint* board)
{
    for (int r = 0; r < N; r++)
    {
        board[R + r] = (Uint)pow(2, N) - 1;
        for (int i = 0; i < N; i++)
        {
            if (board[r * N + i] > 0)
                CLEAR_BIT(board[R + r], (board[r * N + i] - 1));
        }
    }
}

void initializeColumns(Uint* board)
{
    for (int c = 0; c < N; c++)
    {
        board[C + c] = (Uint)pow(2, N) - 1;
        for (int i = 0; i < N; i++)
        {
            if (board[i * N + c] > 0)
                CLEAR_BIT(board[C + c], (board[i * N + c] - 1));
        }
    }
}

void initializeSubboards(Uint* board)
{
    for (int s = 0; s < N; s++)
    {
        board[S + s] = (Uint)pow(2, N) - 1;
        int start = ((s / 3) * 3) * N + ((s % 3) * 3);
        for (int i = 0; i < N; i++)
        {
            int r = i / 3;
            int c = i % 3;
            if (board[start + r * N + c] > 0)
                CLEAR_BIT(board[S + s], (board[start + r * N + c] - 1));
        }
    }
}

Uint* loadBoard()
{
    FILE* boardFile = fopen(BOARD_FILE, "r");
    Uint* board = (Uint*)malloc(N * N * sizeof(Uint));
    char c;

    if (boardFile == NULL)
    {
        printf("boardFile error\n");
        return board;
    }

    for (int i = 0; i < N; i++) 
    {
        for (int j = 0; j < N; j++) 
        {
            if (!fscanf(boardFile, "%c\n", &c)) 
            {
                printf("boardFile error\n");
                return board;
            }

            if (c >= '1' && c <= '9') {
                board[i * N + j] = (Uint)(c - '0');
            }
            else {
                board[i * N + j] = 0;
            }
        }
    }
    return board;
}